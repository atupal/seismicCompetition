#include "hip/hip_runtime.h"
#include <stdio.h>
#include <mex.h>
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define min(ELE1,ELE2) (ELE1)>(ELE2)?(ELE2):(ELE1);
#define GPUMALLOC(POINT,TYPE,SIZE) checkCudaErrors(hipMalloc((void**)&(POINT),sizeof(TYPE)*(SIZE)))
#define GPUMEMCPY(PON1,PON2,TYPE,SIZE,CPYDERICT) checkCudaErrors(hipMemcpy(PON1,PON2,sizeof(TYPE)*(SIZE),CPYDERICT))
typedef double * pdouble;



void rtm2d_fm2d(double *v, double *data,double *boundary, double* M, int nz, int nx, int nt, double dt, double dx);

void mexFunction(int plhs, mxArray *alhs[], const int prhs, 
    const mxArray *arhs[])
{
	int nt = mxGetN(arhs[1]);
	int nz = mxGetM(arhs[0]);
	int nx = mxGetN(arhs[0]);
	int cz = 3;
	double boundary[] = { 4.4385826077685947e-01,
		4.8239114011512579e-01, 5.2191528162939682e-01,
		5.6214244519682266e-01, 6.0275166475009512e-01,
		6.4339270657295611e-01, 6.8369046522894072e-01,
		7.2325024237984237e-01, 7.6166382133052701e-01,
		7.9851621875937684e-01, 8.3339296612850211e-01,
		8.6588774805920488e-01, 8.9561020475518838e-01,
		9.2219369144460783e-01, 9.4530278065205997e-01,
		9.6464029348312264e-01, 9.7995365426708503e-01,
		9.9104037877288398e-01, 9.9775252935263015e-01,
		1.0000000000000000e+00 };
	double * data = mxGetPr(arhs[1]);
	double * v = mxGetPr(arhs[0]);
	double  dt = mxGetPr(arhs[3])[0];
	double  dx = mxGetPr(arhs[2])[0];
	double *temp = NULL;
	alhs[0] = mxCreateDoubleMatrix(nz, nx, mxREAL);
	double * M = mxGetPr(alhs[0]);
	rtm2d_fm2d(v, data,boundary, M, nz, nx, nt, dt, dx);
	
}

void rtm2d_fm2d(double *v , double *data, double *boundary, double* M, int nz, int nx, int nt, double dt, double dx)
{
    pdouble dfdm1,dfdm2,dfdm3,da,db,dM,dsnapshot,ddata,dv,dboundary;
    GPUMALLOC(dfdm1, double, nz*nx);
	GPUMALLOC(dfdm2, double, nz*nx);
	GPUMALLOC(dfdm3, double, nz*nx);
	GPUMALLOC(da, double, nz*nx);
	GPUMALLOC(db, double, nz*nx);
	GPUMALLOC(dM, double, nz*nx);
	GPUMALLOC(dsnapshot, double, nz*nx*nt);
	GPUMALLOC(ddata, double, nt*nx);
	GPUMALLOC(dv, double, nz*nx);
	GPUMALLOC(dboundary, double, 20);

    GPUMEMCPY(dv, v, double, nz*nx, hipMemcpyHostToDevice);
	GPUMEMCPY(dboundary, boundary, double, 20, hipMemcpyHostToDevice);
	GPUMEMCPY(ddata, data, double, nt*nx, hipMemcpyHostToDevice);

    dim3 threadsPerBlocks(16,16);
    dim3 numBlocks(16,16);
    getAB<<<numBlocks, threadsPerBlocks>>>(dv,da,db,dfdm1,dfdm2,
            dfdm3,dM,ddata,dt,dx,nz,nx,nt);

	int cz = 3;
	int bz = 0;
	for (int it = nt - 2; it >= 0; it--)
	{
		cz++;
		bz = min(cz, nz);
        rtm2d_kernel<<<numBlocks,threadsPerBlocks>>>(dfmd1,dfdm2,dfdm3,da,db,
                dboundary,dsnapshot,nz, nx, nt, dx, dt, it,bz);
    }

    rtm2d_to_fm2d<<<numBlocks,threadsPerBlocks>>>(dfdm1,dfdm2,dfdm3,nz,nx);

    for (int it = 1; it < nt; it++)
    {

        fm2d_kernel<<<numBlocks,threadsPerBlocks>>>(dfdm1,dfdm2,dfdm3,da,db,dboundary,dsnapshot,dM,nz,
                nx,nt,dx,dt,it,bz);
    }

    GPUMEMCPY(M,dM,double,nx*nz,hipMemcpyDeviceToHost);
    checkCudaErrors(hipFree(dM));
    checkCudaErrors(hipFree(dv));
    checkCudaErrors(hipFree(dsnapshot));
    checkCudaErrors(hipFree(ddata));
    checkCudaErrors(hipFree(dfdm3));
    checkCudaErrors(hipFree(dfdm2));
    checkCudaErrors(hipFree(dfdm1));
    checkCudaErrors(hipFree(db));
    checkCudaErrors(hipFree(da));
    checkCudaErrors(hipFree(dboundary));
    return ;



}
__global__ void rtm2d_to_fm2d(double *fdm1,double *fdm2,double *fdm3,int nz,int nx)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int idxDim = blockDim.x*gridDim.x;
	int idyDim = blockDim.y*gridDim.y;
    
	for (int ix = idx; ix < nx; ix += idxDim)
	{
		for (int iz = idy; iz < nz; iz += idyDim)
        {
            fdm2[ix*nz+iz] = fdm1[ix*nz+iz];
            fdm1[ix*nz+iz] = 0;
            fdm3[ix*nz+iz] = 0;
        }
    }

}
__global__ void getAB(double *v, double *a, double *b,double *fdm1,double *fdm2,
        double *fdm3,double *M,double * data,double dt,double dx, int nz, int nx,int nt)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int idxDim = blockDim.x*gridDim.x;
	int idyDim = blockDim.y*gridDim.y;
	double tmp;
	for (int ix = idx; ix < nx; ix += idxDim)
	{
		for (int iz = idy; iz < nz; iz += idyDim)
		{
			tmp = v[ix*nz + iz] * dt / dx;
			a[ix*nz + iz] = tmp*tmp;
			b[ix*nz + iz] = 2 - 4.0*a[ix*nz + iz];
            if(iz==0)
            {
                fdm1[ix*nz] = data[(nt - 1)*nx + ix];
                fdm2[ix*nz] = data[(nt - 2)*nx + ix];
                fdm3[ix*nz] = data[(nt - 3)*nx + ix];
            }
            else
            {
                fdm1[ix*nz+iz] =0; 
                fdm2[ix*nz+iz] =0;
                fdm3[ix*nz+iz] =0;
            }
            M[ix*nz+iz]=0;

		}
	}
    
}


__global__ void rtm2d_kernel(double *fmd1,double *fdm2,double*fdm3,double*a,double*b,
        double*boundary,double*snapshot,int nz,int nx,int nt,double dx,double dt,int it,int bz)
{
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    int idz=threadIdx.y+blockIdx.y*blockDim.y;
    int idxDim = blockDim.x*gridDim.x;
    int idzDim = blockDim.y*gridDim.y;

    int ez = (nz == bz) ? nz - 1 : bz;
    int ntt = nx*nz*it;

    for(int ix=idx;ix<nx;ix+=idxDim)
    {
        for(int iz=idz;iz<nz;iz+=idzDim)
        {
            if(ix<20&&iz<bz)
            {
                fdm1[ix*nz + iz] = boundary[ix] * fdm1[ix*nz + iz];
                fdm2[ix*nz + iz] = boundary[ix] * fdm2[ix*nz + iz];
                fdm1[(nx - ix - 1)*nz + iz] = boundary[ix] * fdm1[(nx - ix - 1)*nz + iz];
                fdm2[(nx - ix - 1)*nz + iz] = boundary[ix] * fdm2[(nx - ix - 1)*nz + iz];
            }
            if (bz >= (nz - 19)&&iz>=nz-20&&iz<bz)
            {
                fdm1[ix*nz + iz] = boundary[nz - 1 - iz] * fdm1[ix*nz + iz];
                fdm2[ix*nz + iz] = boundary[nz - 1 - iz] * fdm2[ix*nz + iz];
            }
            if(ix>0&&ix<nx-1&&iz<bz)
            {
                fdm3[ix*nz + iz] -= fdm1[ix*nz + iz];
            }
            if(ix>0&&ix<nx-1&&iz>0&&iz<ez)
			{
				fdm2[ix*nz + iz] = b[ix*nz + iz] * fdm1[ix*nz + iz] + fdm2[ix*nz + iz]
					+ a[(ix + 1)*nz + iz] * fdm1[(ix + 1)*nz + iz]
					+ a[(ix - 1)*nz + iz] * fdm1[(ix - 1)*nz + iz]
					+ a[ix*nz + iz + 1] * fdm1[ix*nz + iz + 1]
					+ a[ix*nz + iz - 1] * fdm1[ix*nz + iz - 1];
			}
            if(ix>0&&ix<nx-1&&iz==0)
            {
                fdm2[ix*nz] = b[ix*nz] * fdm1[ix*nz] + fdm2[ix*nz]
                    + a[(ix + 1)*nz] * fdm1[(ix + 1)*nz]
                    + a[(ix - 1)*nz] * fdm1[(ix - 1)*nz]
                    + a[ix*nz + 1] * fdm1[ix*nz + 1];
            }
            if (bz == nz)
            {
                if(ix>0&&ix<nx-1&&iz==nz-1)
                {
                    for (int ix = 1; ix < nx - 1; ix++)
                        fdm2[ix*nz + nz - 1] = b[ix*nz + nz - 1] * fdm1[ix*nz + nz - 1] + fdm2[ix*nz + nz - 1]
                            + a[(ix + 1)*nz + nz - 1] * fdm1[(ix + 1)*nz + nz - 1]
                            + a[(ix - 1)*nz + nz - 1] * fdm1[(ix - 1)*nz + nz - 1]
                            + a[ix*nz + nz - 2] * fdm1[ix*nz + nz - 2];
                }
                if(ix==0&&iz==nz-1)
                {
                    fdm2[nz - 1] = b[nz - 1] * fdm1[nz - 1] + fdm2[nz - 1]
                        + a[nz + nz - 1] * fdm1[nz + nz - 1] + a[nz - 2] * fdm1[nz - 2];
                }
            }
            if(iz>0&&iz<ez)
            {
                if(ix==0)
                    fdm2[iz] = b[iz] * fdm1[iz] + fdm2[iz]
                        + a[nz + iz] * fdm1[nz + iz]
                        + a[iz + 1] * fdm1[iz + 1]
                        + a[iz - 1] * fdm1[iz - 1];
                if(ix==nx-1)
                    fdm2[(nx - 1)*nz + iz] = b[(nx - 1)*nz + iz] * fdm1[(nx - 1)*nz + iz] + fdm2[(nx - 1)*nz + iz]
                        + a[(nx - 2)*nz + iz] * fdm1[(nx - 2)*nz + iz]
                        + a[(nx - 1)*nz + iz + 1] * fdm1[(nx - 1)*nz + iz + 1]
                        + a[(nx - 1)*nz + iz - 1] * fdm1[(nx - 1)*nz + iz - 1];
            }
		// time extrapolation at corner (0,0)
            if(ix==0&&iz==0)
            {
                fdm2[0] = b[0] * fdm1[0] + fdm2[0]
                    + a[nz] * fdm1[nz] + a[1] * fdm1[1];
            }
		// time extrapolation at corner (0,nx-1)
            if(ix==nx-1&&iz==0)
            {
                fdm2[(nx - 1)*nz] = b[(nx - 1)*nz] * fdm1[(nx - 1)*nz] + fdm2[(nx - 1)*nz]
                    + a[(nx - 2)*nz] * fdm1[(nx - 2)*nz]
                    + a[(nx - 1)*nz + 1] * fdm1[(nx - 1)*nz + 1];
            }
            fdm1[ix*nz+iz]=fdm2[ix*nz+iz];
            fdm2[ix*nz+iz]=fdm3[ix*nz+iz];


            if (it > 1)
            {
                if(iz==0)
                {
                    fdm3[ix*nz] = data[(it - 2)*nx + ix];
                }
                else
                {
                    fdm3[ix*nz]=0.0;
                }
            }

            snapshot[ntt + ix*nz + iz] = fdm1[ix*nz + iz];
        }
    }
}

fdm2[ix*nz+iz]=fdm1[ix*nz+iz];
fdm1[ix*nz+iz]=0.0;
fdm3[ix*nz+iz]=0.0;




__global__ void fm2d_kernel(double *fmd1,double *fdm2,double*fdm3,double*a,double*b,
        double*boundary,double*snapshot,double*M,int nz,int nx,int nt,double dx,double dt,int it,int bz)
{
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    int idz=threadIdx.y+blockIdx.y*blockDim.y;
    int idxDim = blockDim.x*gridDim.x;
    int idzDim = blockDim.y*gridDim.y;

    int tempPoint = (nt-it-1)*nx*nz;

    for(int ix=idx;ix<nx;ix+=idxDim)
    {
        for(int iz=idz;iz<nz;iz+=idzDim)
        {
            if(ix>0&&ix<nx-1&&iz>0&&iz<nz-1)
            {
				fdm3[ix*nz + iz] = b[ix*nz + iz] * fdm2[ix*nz + iz] - fdm1[ix*nz + iz] + \
					a[ix*nz + iz] * (fdm2[(ix + 1)*nz + iz] + fdm2[(ix - 1)*nz + iz] + \
					fdm2[ix*nz + (iz + 1)] + fdm2[ix*nz + (iz - 1)]);
			}

            if(iz>0&&iz<nz-1&&ix==0)
            {
                fdm3[iz] = b[iz] * fdm2[iz] - fdm1[iz] +
                    a[iz] * (fdm2[nz + iz] + fdm2[(iz + 1)] + fdm2[(iz - 1)]);
            }
            if(iz>0&&iz<nz-1&&ix==nx-1)
            {
                fdm3[(nx - 1)*nz + iz] = b[(nx - 1)*nz + iz] * fdm2[(nx - 1)*nz + iz] - fdm1[(nx - 1)*nz + iz] +
                    a[(nx - 1)*nz + iz] * (fdm2[(nx - 2)*nz + iz] + fdm2[(nx - 1)*nz + iz + 1]+ 
                    fdm2[(nx - 1)*nz + iz - 1]);
            }
            if(iz==0&&ix<nx-1&&ix>0)
            {
                fdm3[ix*nz] = b[ix*nz] * fdm2[ix*nz] - fdm1[ix*nz] +
                    a[ix*nz] * (fdm2[ix*nz + 1] + fdm2[(ix + 1)*nz] + fdm2[(ix - 1)*nz]);
            }
            if(iz==nz-1&&ix<nx-1&&ix>0)
            {
                fdm3[ix*nz + (nz - 1)] = b[ix*nz + nz - 1] * fdm2[ix*nz + nz - 1] - fdm1[ix*nz + nz - 1] +
                    a[ix*nz + nz - 1] * (fdm2[ix*nz + (nz - 2)] + fdm2[(ix + 1)*nz + nz - 1] +
                            fdm2[(ix - 1)*nz + nz - 1]);
            }
            if(iz==0&&ix==0)
            {
                fdm3[0] = b[0] * fdm2[0] - fdm1[0] + a[0] * (fdm2[1] + fdm2[nz]);
            }
            if(iz==nz-1&&ix==0)
            {
                fdm3[nz - 1] = b[nz - 1] * fdm2[nz - 1] - fdm1[nz - 1] +
                    a[nz - 1] * (fdm2[nz + nz - 1] + fdm2[nz - 2]);
            }
            if(ix==nx-1&&iz==0)
            {
                fdm3[(nx - 1)*nz] = b[(nx - 1)*nz] * fdm2[(nx - 1)*nz] - fdm1[(nx - 1)*nz] +
                    a[(nx - 1)*nz] * (fdm2[(nx - 2)*nz] + fdm2[(nx - 1)*nz + 1]);
            }
            if(ix==nx-1&&iz==nz-1)
            {
                fdm3[(nx - 1)*nz + (nz - 1)] = b[(nx - 1)*nz + (nz - 1)] * fdm2[(nx - 1)*nz + (nz - 1)] - fdm1[(nx - 1)*nz + (nz - 1)] +
                    a[(nx - 1)*nz + (nz - 1)] * (fdm2[(nx - 1)*nz + (nz - 2)] + fdm2[(nx - 2)*nz + (nz - 1)]);
            }

            fdm1[ix*nz+iz]=fdm2[ix*nz+iz];
            fdm2[ix*nz+iz]=fdm3[ix*nz+iz];

            if(ix<20&&iz<nz-20)
            {
				fdm1[ix*nz + iz] = boundary[ix] * fdm1[ix*nz + iz];
				fdm2[ix*nz + iz] = boundary[ix] * fdm2[ix*nz + iz];
			}
            if(iz<nz-20&&ix>=nx-20&&ix<nx)
            {
				fdm1[ix*nz + iz] = boundary[nx - 1 - ix] * fdm1[ix*nz + iz];
				fdm2[ix*nz + iz] = boundary[nx - 1 - ix] * fdm2[ix*nz + iz];
			}

		
            if(iz>=nz-20&&iz<nz)
            {
				fdm1[ix*nz + iz] = boundary[nz - 1 - iz] * fdm1[ix*nz + iz];
				fdm2[ix*nz + iz] = boundary[nz - 1 - iz] * fdm2[ix*nz + iz];
			}

            M[ix*nz + iz] += fdm1[ix*nz + iz]*snapshot[ix*nz + iz + tempPoint];
        }
	}
}
