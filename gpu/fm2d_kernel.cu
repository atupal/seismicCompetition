
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16 // 32x32 grid (1024 threads total)

__global__ void fm2d_kernel(double* fdm1, double *fdm2, double *fdm3,
        double *boundary, double *a, double *b,
        const int nz, const int nx) {
    int ixt = blockIdx.y * blockDim.y + threadIdx.y; //surface dim [x]
    int izt = blockIdx.x * blockDim.x + threadIdx.x; //depth dim [z]
    int ix = ixt;
    int iz = izt;
    int xg = nx/BLOCK_SIZE; // number of windows in x direction
    int zg = nz/BLOCK_SIZE; // number of windows in z direction
    
    if (nx%BLOCK_SIZE > 0) xg = xg+1;
    if (nz%BLOCK_SIZE > 0) zg = zg+1;
    
    // advance the region of interest
    for (int i = 0; i < zg; i++) {
        iz = izt + (BLOCK_SIZE)*i;
        for (int j = 0; j < xg; j++) {
            ix = ixt + (BLOCK_SIZE)*j;
            
            // finite differencing on interior
            //if (iz > 0 && ix > 0 && iz < nz-2 && ix < nx-2) {
            if (iz+1<nz && ix+1<nx && iz-1>=0 && ix-1>=0){
                fdm3[ix*nz+iz] = b[ix*nz+iz]*fdm2[ix*nz+iz]- fdm1[ix*nz+iz] +
                       a[ix*nz+iz]*(fdm2[(ix+1)*nz+iz] + fdm2[(ix-1)*nz+iz] +
                        fdm2[ix*nz+(iz+1)] + fdm2[ix*nz+(iz-1)]);
            }
            
            // finite differencing at ix = 0
            if (ix == 0 && iz < nz) {
                fdm3[iz] = b[iz]*fdm2[iz] - fdm1[iz] +
                        a[iz]*(fdm2[nz+iz] + fdm2[(iz+1)] + fdm2[(iz-1)]);
            }
            
            // finite differencing at ix = nx-1
            if (ix == nx-1 && iz < nz) {
                fdm3[(nx-1)*nz+iz] = b[(nx-1)*nz+iz]*fdm2[(nx-1)*nz+iz] - fdm1[(nx-1)*nz+iz] +
                        a[(nx-1)*nz+iz]*(fdm2[(nx-2)*nz+iz] + fdm2[(nx-1)*nz+iz+1]
                        + fdm2[(nx-1)*nz+iz-1]);
            }
           
            // finite differencing at iz = 0
            if (iz == 0 && ix < nx) {
                fdm3[ix*nz] = b[ix*nz]*fdm2[ix*nz] -  fdm1[ix*nz] +
                        a[ix*nz]*(fdm2[ix*nz+1] + fdm2[(ix+1)*nz] + fdm2[(ix-1)*nz]);
            }
            
            // finite differencing at iz = nz-1
            if (iz == nz-1 && ix < nx) {
                fdm3[ix*nz+(nz-1)]= b[ix*nz+nz-1]*fdm2[ix*nz+nz-1]- fdm1[ix*nz+nz-1] +
                        a[ix*nz+nz-1]*(fdm2[ix*nz+(nz-2)] + fdm2[(ix+1)*nz+nz-1] +
                        fdm2[(ix-1)*nz+nz-1]);
            }
        }
    }
    // finite differencing at four corners [0][0],[nz-1][0],[0][nx-1],[nz-1][nx-1]
    if (iz == 0 && ix == 0)
        fdm3[0] = b[0]*fdm2[0] -fdm1[0] + a[0]*(fdm2[1] + fdm2[nz]);
    if (iz == nz-1 && ix == 0)
        fdm3[nz-1] = b[nz-1]*fdm2[nz-1] -fdm1[nz-1] + 
                a[nz-1]*(fdm2[nz+nz-1] +fdm2[nz+(nz-2)]);
    if (iz == 0 && ix == nx-1)
        fdm3[(nx-1)*nz] = b[(nx-1)*nz]*fdm2[(nx-1)*nz] -fdm1[(nx-1)*nz] + 
                a[(nx-1)*nz]*(fdm2[(nx-1)*nz] + fdm2[(nx-1)*nz+1]);
    if (iz == nz-1 && ix == nx-1)
        fdm3[(nx-1)*nz+(nz-1)] = b[(nx-1)*nz+(nz-1)]*fdm2[(nx-1)*nz+(nz-1)] -fdm1[(nx-1)*nz+(nz-1)] +
            a[(nx-1)*nz+(nz-1)]*(fdm2[(nx-1)*nz+(nz-2)] +fdm2[(nx-1)*nz+(nz-1)]);
            
    __syncthreads();
    
    for (int i = 0; i < zg; i++) {
        iz = izt + (BLOCK_SIZE)*i;
        for (int j = 0; j < xg; j++) {
            ix = ixt + (BLOCK_SIZE)*j;
            
            // update fdm for next time iteration
            if (iz < nz && ix < nx) {
                fdm1[ix*nz+iz] = fdm2[ix*nz+iz];
                fdm2[ix*nz+iz] = fdm3[ix*nz+iz];
            }
            
            // apply absorbing boundary conditions to 3 sides [not surface]
            if (ix >= 0 && ix < 20){
                fdm1[ix*nz+iz] = boundary[ix]*fdm1[ix*nz+iz];
                fdm2[ix*nz+iz] = boundary[ix]*fdm2[ix*nz+iz];
            }
            
            if (ix >= nx-20 && ix < nx) {
                fdm1[ix*nz+iz] = boundary[nx-1-ix]*fdm1[ix*nz+iz];
                fdm2[ix*nz+iz] = boundary[nx-1-ix]*fdm2[ix*nz+iz];
            }
            
            if (iz >= nz-20 && iz < nz) {
                fdm1[ix*nz+iz] = boundary[nz-1-iz]*fdm1[ix*nz+iz];
                fdm2[ix*nz+iz] = boundary[nz-1-iz]*fdm2[ix*nz+iz];
            }     
       }
    }
}