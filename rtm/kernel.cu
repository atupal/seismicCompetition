#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "gpu/mxGPUArray.h"
#include <stdio.h>
#include <mex.h>
#include "hip/hip_runtime_api.h"

#define min(a,b) (a)>(b)?(b):(a)
#define GPUMALLOC(POINT,TYPE,SIZE) checkCudaErrors(hipMalloc((void**)&(POINT),sizeof(TYPE)*(SIZE)))
#define GPUMEMCPY(PON1,PON2,TYPE,SIZE,CPYDERICT) checkCudaErrors(hipMemcpy(PON1,PON2,sizeof(TYPE)*(SIZE),CPYDERICT))
typedef double * pdouble;

void rtm2d_fm2d(double *v, double *data, double *boundary, double* M, int nz, int nx, int nt, double dt, double dx);
void mexFunction(int plhs, mxArray *alhs[], const int prhs,
	const mxArray *arhs[])
{
	int nt = mxGetN(arhs[1]);
	int nz = mxGetM(arhs[0]);
	int nx = mxGetN(arhs[0]);
	double boundary[] = { 4.4385826077685947e-01,
		4.8239114011512579e-01, 5.2191528162939682e-01,
		5.6214244519682266e-01, 6.0275166475009512e-01,
		6.4339270657295611e-01, 6.8369046522894072e-01,
		7.2325024237984237e-01, 7.6166382133052701e-01,
		7.9851621875937684e-01, 8.3339296612850211e-01,
		8.6588774805920488e-01, 8.9561020475518838e-01,
		9.2219369144460783e-01, 9.4530278065205997e-01,
		9.6464029348312264e-01, 9.7995365426708503e-01,
		9.9104037877288398e-01, 9.9775252935263015e-01,
		1.0000000000000000e+00 };
	double * data = mxGetPr(arhs[1]);
	double * v = mxGetPr(arhs[0]);
	double  dt = mxGetPr(arhs[3])[0];
	double  dx = mxGetPr(arhs[2])[0];
	alhs[0] = mxCreateDoubleMatrix(nz, nx, mxREAL);
	double * M = mxGetPr(alhs[0]);
	rtm2d_fm2d(v, data, boundary, M, nz, nx, nt, dt, dx);

}

__global__ void initialToZero(double * initArray, int size)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int gridSize = blockDim.x*gridDim.x;
	for (int i = id; i < size; i += gridSize)
	{
		initArray[i] = 0.0;
	}
}

__global__ void rtm2dInitialFdm(double*fdm1, double*fdm2, double*fdm3, double*data, int nz, int nx, int nt)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	int gridSize = blockDim.x*gridDim.x;
	for (int ix = id; ix < nx; ix+=gridSize)
	{
		fdm1[ix*nz] = data[(nt - 1)*nx + ix];
		fdm2[ix*nz] = data[(nt - 2)*nx + ix];
		fdm3[ix*nz] = data[(nt - 3)*nx + ix];
	}
	
}

__global__ void getAB(double *v, double *a, double *b,double dt,double dx, int nz, int nx)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int idxDim = blockDim.x*gridDim.x;
	int idyDim = blockDim.y*gridDim.y;
	double tmp;
	for (int ix = idx; ix < nx; ix += idxDim)
	{
		for (int iz = idy; iz < nz; iz += idyDim)
		{
			tmp = v[ix*nz + iz] * dt / dx;
			a[ix*nz + iz] = tmp*tmp;
			b[ix*nz + iz] = 2 - 4.0*a[ix*nz + iz];
		}
	}
}


__global__ void rtm2d_first(double *fdm1, double *fdm2, double *fdm3, double *a, double *b,
	double *boundary, double * snapshot, double *data, int nz, int nx, int nt,int bz,int it)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idz = threadIdx.y + blockIdx.y*blockDim.y;
	int idxDim = blockDim.x*gridDim.x;
	int idzDim = blockDim.y*gridDim.y;
	int tmp = min(bz, nz - 20);
	for (int ix = idx; ix < 20; ix+=idxDim)
	{
		for (int iz = idz; iz < tmp; iz+=idzDim)
		{
			fdm1[ix*nz + iz] = boundary[ix] * fdm1[ix*nz + iz];
			fdm2[ix*nz + iz] = boundary[ix] * fdm2[ix*nz + iz];
			fdm1[(nx - ix - 1)*nz + iz] = boundary[ix] * fdm1[(nx - ix - 1)*nz + iz];
			fdm2[(nx - ix - 1)*nz + iz] = boundary[ix] * fdm2[(nx - ix - 1)*nz + iz];
		}
	}
	if (bz >= (nz - 19))
	{
		for (int ix = idx; ix < nx; ix += idxDim)
		{
			for (int iz = idz+nz - 20; iz < bz; iz+=idzDim)
			{
				if (ix < 20)
				{
					fdm1[ix*nz + iz] = boundary[ix] * fdm1[ix*nz + iz];
					fdm2[ix*nz + iz] = boundary[ix] * fdm2[ix*nz + iz];
					fdm1[(nx - ix - 1)*nz + iz] = boundary[ix] * fdm1[(nx - ix - 1)*nz + iz];
					fdm2[(nx - ix - 1)*nz + iz] = boundary[ix] * fdm2[(nx - ix - 1)*nz + iz];
				}
				fdm1[ix*nz + iz] = boundary[nz - 1 - iz] * fdm1[ix*nz + iz];
				fdm2[ix*nz + iz] = boundary[nz - 1 - iz] * fdm2[ix*nz + iz];
			}
		}
	}
}


__global__ void rtm2d_second(double *fdm1, double *fdm2, double *fdm3, double *a, double *b,
	double *boundary, double * snapshot, double *data, int nz, int nx, int nt, int bz,int it)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idz = threadIdx.y + blockIdx.y*blockDim.y;
	int idxDim = blockDim.x*gridDim.x;
	int idzDim = blockDim.y*gridDim.y;
	int ez = (nz == bz) ? nz - 1 : bz;

	for (int ix = idx+1; ix < nx - 1; ix+=idxDim)
	{
		for (int iz = idz; iz < bz; iz+=idzDim)
		{
			fdm3[ix*nz + iz] -= fdm1[ix*nz + iz];
		}
	}
	for (int ix = idx+1; ix < nx - 1; ix+=idxDim)
	{
		for (int iz = idz+1; iz < ez; iz+=idzDim)
		{
			fdm2[ix*nz + iz] = b[ix*nz + iz] * fdm1[ix*nz + iz] + fdm2[ix*nz + iz]
				+ a[(ix + 1)*nz + iz] * fdm1[(ix + 1)*nz + iz]
				+ a[(ix - 1)*nz + iz] * fdm1[(ix - 1)*nz + iz]
				+ a[ix*nz + iz + 1] * fdm1[ix*nz + iz + 1]
				+ a[ix*nz + iz - 1] * fdm1[ix*nz + iz - 1];
		}
	}
	for (int ix = idz*idxDim+idx+1; ix < nx - 1; ix+=idxDim*idzDim)
	{

		fdm2[ix*nz] = b[ix*nz] * fdm1[ix*nz] + fdm2[ix*nz]
			+ a[(ix + 1)*nz] * fdm1[(ix + 1)*nz]
			+ a[(ix - 1)*nz] * fdm1[(ix - 1)*nz]
			+ a[ix*nz + 1] * fdm1[ix*nz + 1];
	}
	if (bz == nz)
	{
		for (int ix = idz*idxDim+idx+1; ix < nx - 1; ix+=idxDim*idzDim)
			fdm2[ix*nz + nz - 1] = b[ix*nz + nz - 1] * fdm1[ix*nz + nz - 1] + fdm2[ix*nz + nz - 1]
			+ a[(ix + 1)*nz + nz - 1] * fdm1[(ix + 1)*nz + nz - 1]
			+ a[(ix - 1)*nz + nz - 1] * fdm1[(ix - 1)*nz + nz - 1]
			+ a[ix*nz + nz - 2] * fdm1[ix*nz + nz - 2];
		if (idx + idz == 0)
		{
			fdm2[nz - 1] = b[nz - 1] * fdm1[nz - 1] + fdm2[nz - 1]
				+ a[nz + nz - 1] * fdm1[nz + nz - 1] + a[nz - 2] * fdm1[nz - 2];
		}
	}

	for (int iz = idz*idxDim+idx+1; iz < ez; iz+=idxDim*idzDim)
	{
		// time extrapolation at ix = 0
		fdm2[iz] = b[iz] * fdm1[iz] + fdm2[iz]
			+ a[nz + iz] * fdm1[nz + iz]
			+ a[iz + 1] * fdm1[iz + 1]
			+ a[iz - 1] * fdm1[iz - 1];
		fdm2[(nx - 1)*nz + iz] = b[(nx - 1)*nz + iz] * fdm1[(nx - 1)*nz + iz] + fdm2[(nx - 1)*nz + iz]
			+ a[(nx - 2)*nz + iz] * fdm1[(nx - 2)*nz + iz]
			+ a[(nx - 1)*nz + iz + 1] * fdm1[(nx - 1)*nz + iz + 1]
			+ a[(nx - 1)*nz + iz - 1] * fdm1[(nx - 1)*nz + iz - 1];
	}
	if (idx + idz == 0)
	{
		// time extrapolation at corner (0,0)
		fdm2[0] = b[0] * fdm1[0] + fdm2[0]
			+ a[nz] * fdm1[nz] + a[1] * fdm1[1];

		// time extrapolation at corner (0,nx-1)
		fdm2[(nx - 1)*nz] = b[(nx - 1)*nz] * fdm1[(nx - 1)*nz] + fdm2[(nx - 1)*nz]
			+ a[(nx - 2)*nz] * fdm1[(nx - 2)*nz]
			+ a[(nx - 1)*nz + 1] * fdm1[(nx - 1)*nz + 1];
	}
}

__global__ void rtm2d_third(double *fdm1, double *fdm2, double *fdm3, double *a, double *b,
	double *boundary, double * snapshot, double *data, int nz, int nx, int nt, int bz,int it)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idz = threadIdx.y + blockIdx.y*blockDim.y;
	int idxDim = blockDim.x*gridDim.x;
	int idzDim = blockDim.y*gridDim.y;
	for (int ix = idx; ix < nx; ix += idxDim)
	{
		for (int iz = idz; iz < nz; iz += idzDim)
		{
			fdm1[ix*nz + iz] = fdm2[ix*nz + iz];
			fdm2[ix*nz + iz] = fdm3[ix*nz + iz];
			if (it > 1)
			{
				if (iz == 0)
					fdm3[ix*nz] = data[(it - 2)*nx + ix];
				else
					fdm3[ix*nz + iz] = 0;
			}
		}
	}
}

__global__ void gpuCopy(double *pot1, double *pot2, int excursion,int size)
{
	
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idxDim = blockDim.x*gridDim.x;
	for (int ix = idx; ix < size; ix += idxDim)
	{
		pot1[ix+excursion] = pot2[ix];
	}
}

__global__ void fm2d_first(double *fdm1, double *fdm2, double *fdm3, double *a, double *b,
	double *boundary, double * snapshot, double *data,double *M, int nz, int nx, int nt, int bz, int it)
{

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idz = threadIdx.y + blockIdx.y*blockDim.y;
	int idxDim = blockDim.x*gridDim.x;
	int idzDim = blockDim.y*gridDim.y;

	for (int ix = idx+1; ix < nx - 1; ix+=idxDim)
	{
		for (int iz = idz+1; iz < nz - 1; iz+=idzDim)
		{
			fdm3[ix*nz + iz] = b[ix*nz + iz] * fdm2[ix*nz + iz] - fdm1[ix*nz + iz] + \
				a[ix*nz + iz] * (fdm2[(ix + 1)*nz + iz] + fdm2[(ix - 1)*nz + iz] + \
				fdm2[ix*nz + (iz + 1)] + fdm2[ix*nz + (iz - 1)]);
		}
	}
	for (int iz = idx+idz*idxDim+1; iz < nz - 1; iz+=idxDim*idzDim)
	{
		fdm3[iz] = b[iz] * fdm2[iz] - fdm1[iz] +
			a[iz] * (fdm2[nz + iz] + fdm2[(iz + 1)] + fdm2[(iz - 1)]);
		fdm3[(nx - 1)*nz + iz] = b[(nx - 1)*nz + iz] * fdm2[(nx - 1)*nz + iz] - fdm1[(nx - 1)*nz + iz] +
			a[(nx - 1)*nz + iz] * (fdm2[(nx - 2)*nz + iz] + fdm2[(nx - 1)*nz + iz + 1]
			+ fdm2[(nx - 1)*nz + iz - 1]);
	}
	for (int ix = idx+idz*idxDim+1; ix < nx - 1; ix+=idxDim*idzDim)
	{
		fdm3[ix*nz] = b[ix*nz] * fdm2[ix*nz] - fdm1[ix*nz] +
			a[ix*nz] * (fdm2[ix*nz + 1] + fdm2[(ix + 1)*nz] + fdm2[(ix - 1)*nz]);
		fdm3[ix*nz + (nz - 1)] = b[ix*nz + nz - 1] * fdm2[ix*nz + nz - 1] - fdm1[ix*nz + nz - 1] +
			a[ix*nz + nz - 1] * (fdm2[ix*nz + (nz - 2)] + fdm2[(ix + 1)*nz + nz - 1] +
			fdm2[(ix - 1)*nz + nz - 1]);
	}
	if (idz + idx == 0)
	{
		//  % finite differencing at four corners (1,1), (nz,1), (1,nx), (nz,nx)
		fdm3[0] = b[0] * fdm2[0] - fdm1[0] + a[0] * (fdm2[1] + fdm2[nz]);
		fdm3[nz - 1] = b[nz - 1] * fdm2[nz - 1] - fdm1[nz - 1] +
			a[nz - 1] * (fdm2[nz + nz - 1] + fdm2[nz - 2]);
		//a[nz - 1] * (fdm2[nz + nz - 1] + fdm2[nz + (nz - 2)]); // BUGBUG

		fdm3[(nx - 1)*nz] = b[(nx - 1)*nz] * fdm2[(nx - 1)*nz] - fdm1[(nx - 1)*nz] +
			a[(nx - 1)*nz] * (fdm2[(nx - 2)*nz] + fdm2[(nx - 1)*nz + 1]);
		//a[(nx - 1)*nz] * (fdm2[(nx - 1)*nz] + fdm2[(nx - 1)*nz + 1]); //BUGBUG
		fdm3[(nx - 1)*nz + (nz - 1)] = b[(nx - 1)*nz + (nz - 1)] * fdm2[(nx - 1)*nz + (nz - 1)] - fdm1[(nx - 1)*nz + (nz - 1)] +
			a[(nx - 1)*nz + (nz - 1)] * (fdm2[(nx - 1)*nz + (nz - 2)] + fdm2[(nx - 2)*nz + (nz - 1)]);
		//a[(nx - 1)*nz + (nz - 1)] * (fdm2[(nx - 1)*nz + (nz - 2)] + fdm2[(nx - 1)*nz + (nz - 1)]); //BUGBUG

	}
	return;
}

__global__ void fm2d_second(double *fdm1, double *fdm2, double *fdm3, double *a, double *b,
	double *boundary, double * snapshot, double *data,double *M, int nz, int nx, int nt, int bz, int it)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idz = threadIdx.y + blockIdx.y*blockDim.y;
	int idxDim = blockDim.x*gridDim.x;
	int idzDim = blockDim.y*gridDim.y;
	int tempPoint = (nt - it - 1)*nx*nz;

	for (int ix = idx; ix < nx; ix += idxDim)
	{
		for (int iz = idz; iz < nz; iz += idzDim)
		{
			fdm1[ix*nz + iz] = fdm2[ix*nz + iz];
			fdm2[ix*nz + iz] = fdm3[ix*nz + iz];
			if (ix < 20 && iz < nz - 20)
			{
				fdm1[ix*nz + iz] = boundary[ix] * fdm1[ix*nz + iz];
				fdm2[ix*nz + iz] = boundary[ix] * fdm2[ix*nz + iz];
			}
			if (ix >= nx - 20 && ix < nx&&iz < nz - 20)
			{
				fdm1[ix*nz + iz] = boundary[nx - 1 - ix] * fdm1[ix*nz + iz];
				fdm2[ix*nz + iz] = boundary[nx - 1 - ix] * fdm2[ix*nz + iz];
			}
			if (iz >= nz - 20 && iz < nz)
			{
				fdm1[ix*nz + iz] = boundary[nz - 1 - iz] * fdm1[ix*nz + iz];
				fdm2[ix*nz + iz] = boundary[nz - 1 - iz] * fdm2[ix*nz + iz];
			}
			M[ix*nz + iz] += fdm1[ix*nz + iz] * snapshot[ix*nz + iz + tempPoint];
		}
	}
	return;
}


void rtm2d_fm2d(double *v, double *data, double *boundary, double* M, int nz, int nx, int nt, double dt, double dx)
{

	int cz = 3;
	int bz = 0;

	pdouble dfdm1, dfdm2, dfdm3, dv,da, db, dM, dsnapshot,ddata,dboundary;
    mxInitGPU();
	GPUMALLOC(dfdm1, double, nz*nx);
	GPUMALLOC(dfdm2, double, nz*nx);
	GPUMALLOC(dfdm3, double, nz*nx);
	GPUMALLOC(da, double, nz*nx);
	GPUMALLOC(db, double, nz*nx);
	GPUMALLOC(dM, double, nz*nx);
	GPUMALLOC(dsnapshot, double, nz*nx*nt);
	GPUMALLOC(ddata, double, nt*nx);
	GPUMALLOC(dv, double, nz*nx);
	GPUMALLOC(dboundary, double, 20);

	initialToZero <<<256, 512 >>>(dfdm1, nz*nx);
	initialToZero <<<256, 512 >>>(dfdm3, nz*nx);
	initialToZero <<<256, 512 >>>(dfdm2, nz*nx);
	initialToZero <<<256, 512 >>>(dM, nz*nx);

	GPUMEMCPY(dv, v, double, nz*nx, hipMemcpyHostToDevice);
	GPUMEMCPY(dboundary, boundary, double, 20, hipMemcpyHostToDevice);

	GPUMEMCPY(ddata, data, double, nt*nx, hipMemcpyHostToDevice);
	rtm2dInitialFdm <<<256, 512 >>>(dfdm1, dfdm2, dfdm3, ddata, nz, nx, nt);

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(16, 16);
	getAB <<<numBlocks, threadsPerBlock >>>(dv, da, db, dt, dx, nz, nx);
 	checkCudaErrors(hipFree(dv));


	for (int it = nt - 2; it >= 0; it--)
	{
		cz++;
		bz = min(cz, nz);

		rtm2d_first <<<numBlocks, threadsPerBlock >>>(dfdm1, dfdm2, dfdm3, da, db, dboundary, dsnapshot,
			ddata, nz, nx, nt, bz, it);
		rtm2d_second <<<numBlocks, threadsPerBlock >>>(dfdm1, dfdm2, dfdm3, da, db, dboundary, dsnapshot,
			ddata, nz, nx, nt, bz, it);
		rtm2d_third <<<numBlocks, threadsPerBlock >>>(dfdm1, dfdm2, dfdm3, da, db, dboundary, dsnapshot,
			ddata, nz, nx, nt, bz, it);


		int ntt = nx*nz*it;
		gpuCopy <<<256, 512 >>>(dsnapshot,dfdm1, ntt, nz*nx);
	}


	gpuCopy <<<256, 512 >>>(dfdm2, dfdm1,0, nz*nx);
	initialToZero <<<256, 512 >>>(dfdm1, nz*nx);
	initialToZero <<<256, 512 >>>(dfdm3, nz*nx);


	//it prablem
	for (int it = 1; it < nt; it++)
	{

		fm2d_first <<<numBlocks, threadsPerBlock >>>(dfdm1, dfdm2, dfdm3, da, db, dboundary,
			dsnapshot, ddata, dM, nz, nx, nt, bz, it);
		fm2d_second <<<numBlocks, threadsPerBlock >>>(dfdm1, dfdm2, dfdm3, da, db, dboundary,
			dsnapshot, ddata, dM, nz, nx, nt, bz, it);

	}
	GPUMEMCPY(M, dM, double, nz*nx, hipMemcpyDeviceToHost);
	checkCudaErrors(hipFree(dM));
	checkCudaErrors(hipFree(dfdm1));
	checkCudaErrors(hipFree(dfdm2));
	checkCudaErrors(hipFree(dfdm3));
	checkCudaErrors(hipFree(da));
	checkCudaErrors(hipFree(db));
	checkCudaErrors(hipFree(dboundary));
	checkCudaErrors(hipFree(dsnapshot));
	checkCudaErrors(hipFree(ddata));
	return;
}
