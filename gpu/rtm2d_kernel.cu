
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16 // 32x32 grid (1024 threads total)

__global__ void rtm2d_kernel(double* fdm1, double *fdm2, double *fdm3,
        double *boundary, double *a, double *b,
        const int nz, const int nx, int bz, int it,
        double *data, const int nt) {
    
    int ixt = blockIdx.y * blockDim.y + threadIdx.y; //surface dim [x]
    int izt = blockIdx.x * blockDim.x + threadIdx.x; //depth dim [z]
    int ix = ixt;
    int iz = izt;
    int ez = bz;
    int xg = nx/BLOCK_SIZE; // number of windows in x direction
    int zg = nz/BLOCK_SIZE; // number of windows in z direction
    
    if (nx%BLOCK_SIZE > 0) xg = xg+1;
    if (nz%BLOCK_SIZE > 0) zg = zg+1;
    
    for (int i = 0; i < zg; i++) {
        iz = izt + (BLOCK_SIZE)*i;
        for (int j = 0; j < xg; j++) {
            ix = ixt + (BLOCK_SIZE)*j;
            
            // apply absorbing boundary condition on left
            if (ix >= 0 && ix < 20 && iz < nz) {
                fdm1[ix*nz+iz] = boundary[ix]*fdm1[ix*nz+iz];
                fdm2[ix*nz+iz] = boundary[ix]*fdm2[ix*nz+iz];
            }
            
            // apply absorbing boundary condition on right
            if (ix >= nx-20 && ix < nx && iz < nz) {
                fdm1[ix*nz+iz] = boundary[nx-1-ix]*fdm1[ix*nz+iz];
                fdm2[ix*nz+iz] = boundary[nx-1-ix]*fdm2[ix*nz+iz];
            }
            
            // apply absorbing boundary condition at depth
            if (bz>=nz-20 && iz >= nz-20 && iz < nz) {
                fdm1[ix*nz+iz] = boundary[nz-1-iz]*fdm1[ix*nz+iz];
                fdm2[ix*nz+iz] = boundary[nz-1-iz]*fdm2[ix*nz+iz];
            }
        } // j loop
    }// i loop
    
    //__syncthreads();
    
    for (int i = 0; i < zg; i++) {
        iz = izt + (BLOCK_SIZE)*i;
        for (int j = 0; j < xg; j++) {
            ix = ixt + (BLOCK_SIZE)*j;
            
            // computing grid depth (extent in z to solve)
            if (bz == nz)
                ez = nz-2;
            else
                ez = bz;
            
            // time extrapolation between iz and bz
            if (iz < bz && ix < nx)
                fdm3[ix*nz+iz] = fdm3[ix*nz+iz] - fdm1[ix*nz+iz];
            
            //time extrapolation over interior
            if(iz > 0 && iz < ez && ix > 0 && ix < nx-1)
                fdm2[ix*nz+iz] = b[ix*nz+iz]*fdm1[ix*nz+iz] + fdm2[ix*nz+iz]
                        + a[(ix+1)*nz+iz]*fdm1[(ix+1)*nz+iz]
                        + a[(ix-1)*nz+iz]*fdm1[(ix-1)*nz+iz]
                        + a[ix*nz+iz+1]*fdm1[ix*nz+iz+1]
                        + a[ix*nz+iz-1]*fdm1[ix*nz+iz-1];
            
            // time extrapolation at iz = 0
            if (iz == 0 && ix > 0 && ix < nx-1)
                fdm2[ix*nz] = b[ix*nz]*fdm1[ix*nz] + fdm2[ix*nz]
                        + a[(ix+1)*nz]*fdm1[(ix+1)*nz]
                        + a[(ix-1)*nz]*fdm1[(ix-1)*nz]
                        + a[ix*nz+1]*fdm1[ix*nz+1];
            
            if (iz > 0 && iz < ez && ix == 0)
                // time extrapolation at ix = 0
                fdm2[iz] = b[iz]*fdm1[iz] + fdm2[iz]
                        + a[nz+iz]*fdm1[nz+iz]
                        + a[iz+1]*fdm1[iz+1]
                        + a[iz-1]*fdm1[iz-1];
            
            if (iz > 0 && iz < ez && ix == nx-1)
                //time extrapolation at ix = nx-1
                fdm2[(nx-1)*nz+iz] = b[(nx-1)*nz+iz]*fdm1[(nx-1)*nz+iz] + fdm2[(nx-1)*nz+iz]
                        + a[(nx-2)*nz+iz]*fdm1[(nx-2)*nz+iz]
                        + a[(nx-1)*nz+iz+1]*fdm1[(nx-1)*nz+iz+1]
                        + a[(nx-1)*nz+iz-1]*fdm1[(nx-1)*nz+iz-1];
            
            if (bz == nz) {
                if (iz == nz-1 && ix > 0  && ix < nx)
                    // time extrapolation at iz = nz-1
                    fdm2[ix*nz+nz-1] = b[ix*nz+nz-1]*fdm1[ix*nz+nz-1] + fdm2[ix*nz+nz-1]
                            + a[(ix+1)*nz+nz-1]*fdm1[(ix+1)*nz+nz-1]
                            + a[(ix-1)*nz+nz-1]*fdm1[(ix-1)*nz+nz-1]
                            + a[ix*nz+nz-2]*fdm1[ix*nz+nz-2];
                
                if (iz == nz-1 && ix == 0)
                    // time extrapolation at corner (nz-1,0)
                    fdm2[nz-1] = b[nz-1]*fdm1[nz-1] + fdm2[nz-1]
                            + a[nz+nz-1]*fdm1[nz+nz-1] + a[nz-2]*fdm1[nz-2];
            }
            
            if (iz == 0 && ix == 0)
                // time extrapolation at corner (0,0)
                fdm2[0] = b[0]*fdm1[0] + fdm2[0]
                        + a[nz]*fdm1[nz] + a[1]*fdm1[1];
            
            if (iz == 0 && ix == nx-1)
                // time extrapolation at corner (0,nx-1)
                fdm2[(nx-1)*nz] = b[(nx-1)*nz]*fdm1[(nx-1)*nz] + fdm2[(nx-1)*nz]
                        + a[(nx-2)*nz]*fdm1[(nx-2)*nz]
                        + a[(nx-1)*nz+1]*fdm1[(nx-1)*nz+1];
        } // j loop
    } // i loop
    
//__syncthreads();
    
    for (int i = 0; i < zg; i++) {
        iz = izt + (BLOCK_SIZE)*i;
        for (int j = 0; j < xg; j++) {
            ix = ixt + (BLOCK_SIZE)*j;
            
            if (ix<nx && iz<nz) {
                // set up fdm for next iteration
                fdm1[ix*nz+iz] = fdm2[ix*nz+iz];
                fdm2[ix*nz+iz] = fdm3[ix*nz+iz];
                
                
                // insert surface boundary wavefield
                if (it > 2) {
                    if (iz>0)    fdm3[ix*nz+iz] = 0;
                    if (iz == 0) fdm3[ix*nz] = data[(it-3)*nx+ix];
                }
            }
            __syncthreads();
        } // j loop
    } // i loop
} //rtm2d_step